// author bufanzhen
// date 20240506
// barrett reduction:
// for input z(128bit),p(64bit)
// output z mod p


#include <iostream>
using namespace std;
#include "hip/hip_runtime.h"


#include <inttypes.h>





// compute miu = floor(b**2k/p)
// __device__ void preCompute(uint32_t P[n],uint32_t miu[2]){
//     return ;
// }

__device__   __forceinline__ void MULT(uint32_t a[],uint32_t b[],uint32_t alen,uint32_t blen,uint32_t res[]){
    uint32_t reslen = alen + blen;
    
    res[reslen-1] = 0;
    asm("mul.lo.u32 %0, %2, %3; mul.hi.u32 %1, %2, %3;"
        : "=r"(res[0]), "=r"(res[1])
        : "r"(a[0]), "r"(b[0]));
    asm("mul.lo.u32 %0, %2, %3; mul.hi.u32 %1, %2, %3;"
        : "=r"(res[2]), "=r"(res[3])
        : "r"(a[1]), "r"(b[1]));
    asm("mad.lo.cc.u32 %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
        : "+r"(res[1]), "+r"(res[2])
        : "r"(a[1]), "r"(b[0]));
    asm("addc.u32 %0, %0, 0;" : "+r"(res[3]));
    asm("mad.lo.cc.u32 %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
        : "+r"(res[1]), "+r"(res[2])
        : "r"(a[0]), "r"(b[1]));
    asm("addc.u32 %0, %0, 0;" : "+r"(res[3]));
    asm("mad.lo.u32.cc %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
        : "+r"(res[2]), "+r"(res[3])
        : "r"(a[2]), "r"(b[0]));
    asm("addc.u32 %0, %0, 0;" : "+r"(res[4]));
    asm("mad.lo.cc.u32 %0, %2, %3, %0; madc.hi.u32 %1, %2, %3, %1;"
        : "=r"(res[3]), "+r"(res[4])
        : "r"(a[2]), "r"(b[1]));
    return ;
}


__device__ __forceinline__ void computeQhat(uint32_t Z[],uint32_t miu[],uint32_t Qhat[]){
    uint32_t tmpZ[3] = {0,0,0};
    tmpZ[0] = Z[2]<<1|Z[1]>>31;
    tmpZ[1] = Z[3]<<1|Z[2]>>31;
    tmpZ[2] = Z[3]>>31;
    uint32_t tmpQhat[4] = {0,0,0,0};
    asm("mul.hi.u32 %0, %1, %2;": "=r"(tmpQhat[0]): "r"(tmpZ[0]), "r"(miu[0]));
    asm("mul.lo.u32 %0, %2, %3; mul.hi.u32 %1, %2, %3;"
        : "=r"(tmpQhat[1]), "=r"(tmpQhat[2])
        : "r"(tmpZ[1]), "r"(miu[1]));
    asm("mad.lo.cc.u32 %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
        : "+r"(tmpQhat[0]), "+r"(tmpQhat[1])
        : "r"(tmpZ[1]), "r"(miu[0]));
    asm("addc.cc.u32 %0, %0, 0;" : "+r"(tmpQhat[2]));
    asm("addc.u32 %0, %0, 0;" : "+r"(tmpQhat[3]));
    asm("mad.lo.cc.u32 %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
        : "+r"(tmpQhat[0]), "+r"(tmpQhat[1])
        : "r"(tmpZ[0]), "r"(miu[1]));
    asm("addc.cc.u32 %0, %0, 0;" : "+r"(tmpQhat[2]));
    asm("addc.u32 %0, %0, 0;" : "+r"(tmpQhat[3]));
    asm("mad.lo.u32.cc %0, %2, %3, %0; madc.lo.cc.u32 %1, %2, %4, %1;"
        : "+r"(tmpQhat[1]), "+r"(tmpQhat[2])
        : "r"(tmpZ[2]), "r"(miu[0]), "r"(miu[1]));
    asm("addc.u32 %0, %0, 0;" : "+r"(tmpQhat[3]));
    asm("mad.lo.cc.u32 %0, %2, %3, %0; madc.lo.cc.u32 %1, %2, %4, %1;"
        : "+r"(tmpQhat[1]), "+r"(tmpQhat[2])
        : "r"(miu[2]), "r"(tmpZ[0]), "r"(tmpZ[1]));
    asm("addc.u32 %0, %0, 0;" : "+r"(tmpQhat[3]));
    asm("mad.lo.u32 %0, %1, %2, %0;"
        : "+r"(tmpQhat[3])
        : "r"(miu[2]), "r"(tmpZ[2]));
    Qhat[0] = tmpQhat[1]>>1|tmpQhat[2]<<31;
    Qhat[1] = tmpQhat[2]>>1|tmpQhat[3]<<31;
    Qhat[2] = tmpQhat[3]>>1;
    return ;
}

__device__ __forceinline__ void computeR2(uint32_t r2[], uint32_t qHat[],uint32_t p[]){
    asm("mul.lo.u32 %0, %2, %3; mul.hi.u32 %1, %2, %3;"
        : "=r"(r2[0]), "=r"(r2[1])
        : "r"(qHat[0]), "r"(p[0]));
    asm("mul.lo.u32 %0, %1, %2;"
        : "=r"(r2[2])
        : "r"(qHat[1]), "r"(p[1]));
    asm("mad.lo.u32.cc %0, %2, %3, %0; madc.hi.u32 %1, %2, %3, %1;"
        : "+r"(r2[1]), "+r"(r2[2])
        : "r"(qHat[1]), "r"(p[0]));
    asm("mad.lo.u32.cc %0, %2, %3, %0; madc.hi.u32 %1, %2, %3, %1;"
        : "+r"(r2[1]), "+r"(r2[2])
        : "r"(qHat[0]), "r"(p[1]));
    asm("mul.lo.u32 %0, %1, %2;"
        : "+r"(r2[2])
        : "r"(qHat[2]), "r"(p[0]));
    r2[2] = r2[2]&0x1;
    return ;
}


__device__ void BarrettRedc(uint32_t Z[],uint32_t p[],uint32_t miu[],uint32_t res[]){
    // compute q_hat
    uint32_t qHat[3] = {0,0,0};
    computeQhat(Z,miu,qHat);

    // compute r
    uint32_t r1[3] = {Z[0],Z[1],Z[2]&0x1};
    uint32_t r2[3] = {0,0,0};
    computeR2(r2,qHat,p);
    

    // compare R1 and R2 compute r=R1-R2
    // if r1<r2 r1=r1+x**(k+1)
    if(r1[2]<r2[2]||((r1[2]==r2[2])&&r1[1]<r2[1])||(((r1[2]==r2[2])&&(r1[1]==r2[1])&&(r1[0]<r2[0])))){
        r1[2]+=1;
    }
    asm("sub.cc.u32 %0, %0, %1;" : "+r"(r1[0]) : "r"(r2[0]));
    asm("subc.cc.u32 %0, %0, %1;" : "+r"(r1[1]) : "r"(r2[1]));
    asm("subc.cc.u32 %0, %0, %1;" : "+r"(r1[2]) : "r"(r2[2]));

    // now r1 is r
    while((r1[2]>0)||(r1[1]>p[1])||((r1[1]==p[1])&&(r1[0]>=p[0]))){
        asm("sub.cc.u32 %0, %0, %1;" : "+r"(r1[0]) : "r"(p[0]));
        asm("subc.cc.u32 %0, %0, %1;" : "+r"(r1[1]) : "r"(p[1]));
        asm("subc.cc.u32 %0, %0, 0;" : "+r"(r1[2]));
    }
    res[1] = r1[1];
    res[0] = r1[0];
    return ;

}

__global__ void func1(uint32_t Z[],uint32_t P[],uint32_t res[]){
    
    uint32_t miu[3] =   {0x708a9a6b,0xb1dbb0f0,0x1};
    BarrettRedc(Z,P,miu,res);
    return ;
}


int main(){

    uint32_t Plen = 64;
    uint32_t P[2]   =   {0x34f7fbb5,0x970dd129};
    uint32_t Z[4] =   {0xb694bf5b,0xe90cbb0e,0xf6b92885,0xde43d922};



    uint32_t *d_Z,*d_P,*d_res;
    uint32_t *res = new uint32_t[2];
    hipMalloc((void**)&d_P , sizeof(uint32_t) * 2);
    hipMalloc((void**)&d_Z , sizeof(uint32_t) * 4);
    hipMalloc((void**)&d_res , sizeof(uint32_t) * 2);
    hipMemcpy( d_P,P , sizeof(uint32_t) * 2, hipMemcpyHostToDevice);
    hipMemcpy( d_Z,Z , sizeof(uint32_t) * 4, hipMemcpyHostToDevice);

    
    func1<<<1,32>>>(d_Z,d_P,d_res);


    hipMemcpy( res, d_res, sizeof(uint32_t) * 2, hipMemcpyDeviceToHost);
    for(int i = 0;i<2;i++){
        cout<<std::hex<<res[i]<<endl;
    }
    return 0;
}